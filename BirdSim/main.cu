#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define MAX_BIRDS 10000
#define EPSILON 1e-6f
#define PERCEPTION_RADIUS 15.0f
#define SEPARATION_RADIUS_FACTOR 0.3f
#define MAX_SPEED 3.0f
#define MAX_FORCE 0.1f
#define CELL_SIZE 30.0f

typedef unsigned char uchar;

// Performance tracking structure
struct PerformanceMetrics {
    float gridUpdateTime;
    float forceCalculationTime;
    float positionUpdateTime;
    int stepsCompleted;
    float totalTime;
};

// Bird data structure
struct Bird {
    float3 position;
    float3 velocity;
    float3 acceleration;
    int dominantForce; // 0 = separation, 1 = alignment, 2 = cohesion
};

// Spatial grid for optimization
struct SpatialGrid {
    int* cells;
    int* cellStartIndices;
    int* cellEndIndices;
    int* particleIndices;
    int gridSizeX;
    int gridSizeY;
    int gridSizeZ;
    float cellSize;
    float3 minBounds;
};

// Global variables for the simulation
__device__ __managed__ Bird birds[MAX_BIRDS];
__device__ __managed__ int numBirds;
__device__ __managed__ float3 minBounds;
__device__ __managed__ float3 maxBounds;
__device__ __managed__ SpatialGrid grid;
__device__ __managed__ PerformanceMetrics metrics;

// CUDA resources
hiprandState* d_states = nullptr;
int* d_cellIndices = nullptr;
int* d_particleIndices = nullptr;
int* d_cellStartIndices = nullptr;
int* d_cellEndIndices = nullptr;
hipEvent_t startEvent, stopEvent, gridStartEvent, gridStopEvent, forceStartEvent, forceStopEvent, posStartEvent, posStopEvent;

// Initialize random number generator
__global__ void setupRNG(hiprandState* states) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numBirds) {
        hiprand_init(clock64(), tid, 0, &states[tid]);
    }
}

// Initialize birds with random positions and velocities
__global__ void initBirds(hiprandState* states) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numBirds) {
        hiprandState localState = states[tid];

        birds[tid].position.x = minBounds.x + hiprand_uniform(&localState) * (maxBounds.x - minBounds.x);
        birds[tid].position.y = minBounds.y + hiprand_uniform(&localState) * (maxBounds.y - minBounds.y);
        birds[tid].position.z = minBounds.z + hiprand_uniform(&localState) * (maxBounds.z - minBounds.z);

        float3 vel;
        vel.x = hiprand_uniform(&localState) * 2.0f - 1.0f;
        vel.y = hiprand_uniform(&localState) * 2.0f - 1.0f;
        vel.z = hiprand_uniform(&localState) * 2.0f - 1.0f;

        float len = sqrtf(vel.x * vel.x + vel.y * vel.y + vel.z * vel.z);
        if (len > EPSILON) {
            vel.x /= len;
            vel.y /= len;
            vel.z /= len;
        }

        float speed = 0.5f + hiprand_uniform(&localState) * 1.5f;
        birds[tid].velocity = make_float3(vel.x * speed, vel.y * speed, vel.z * speed);
        birds[tid].acceleration = make_float3(0.0f, 0.0f, 0.0f);
        birds[tid].dominantForce = 0;

        states[tid] = localState;
    }
}

// Calculate the cell coordinates for a given position
__device__ int3 getCellCoords(float3 position, float3 minBounds, float cellSize, int gridSizeX, int gridSizeY, int gridSizeZ) {
    int x = (int)floorf((position.x - minBounds.x) / cellSize);
    int y = (int)floorf((position.y - minBounds.y) / cellSize);
    int z = (int)floorf((position.z - minBounds.z) / cellSize);

    x = max(0, min(gridSizeX - 1, x));
    y = max(0, min(gridSizeY - 1, y));
    z = max(0, min(gridSizeZ - 1, z));

    return make_int3(x, y, z);
}

// Calculate flat index for 3D grid
__device__ int getCellIndex(int3 cellCoords, int gridSizeX, int gridSizeY) {
    return cellCoords.z * gridSizeX * gridSizeY + cellCoords.y * gridSizeX + cellCoords.x;
}

// Reset the spatial grid
__global__ void resetGrid(int* cellStartIndices, int* cellEndIndices, int cellCount) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < cellCount) {
        cellStartIndices[tid] = -1;
        cellEndIndices[tid] = -1;
    }
}

// Calculate cell indices for all birds
__global__ void calculateCellIndices(int* particleIndices, int* cells) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numBirds) {
        int3 cellCoords = getCellCoords(
            birds[tid].position,
            grid.minBounds,
            grid.cellSize,
            grid.gridSizeX,
            grid.gridSizeY,
            grid.gridSizeZ
        );

        int cellIndex = getCellIndex(cellCoords, grid.gridSizeX, grid.gridSizeY);
        cells[tid] = cellIndex;
        particleIndices[tid] = tid;
    }
}

// Sort birds by cell index (using simple counting sort for simplicity)
__global__ void countCellElements(int* cells, int* cellStartIndices, int* cellEndIndices) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numBirds) {
        int cellIndex = cells[tid];

        // Use atomicMin/Max for thread safety
        atomicMin(&cellStartIndices[cellIndex], tid);
        atomicMax(&cellEndIndices[cellIndex], tid);
    }
}

// Calculate the wrapped distance between two positions
__device__ float3 calculateWrappedDistance(float3 pos1, float3 pos2) {
    float3 diff = make_float3(pos1.x - pos2.x, pos1.y - pos2.y, pos1.z - pos2.z);

    float size_x = maxBounds.x - minBounds.x;
    float size_y = maxBounds.y - minBounds.y;
    float size_z = maxBounds.z - minBounds.z;

    // Handle wraparound (toroidal space)
    if (abs(diff.x) > size_x * 0.5f) {
        diff.x = diff.x - copysignf(size_x, diff.x);
    }
    if (abs(diff.y) > size_y * 0.5f) {
        diff.y = diff.y - copysignf(size_y, diff.y);
    }
    if (abs(diff.z) > size_z * 0.5f) {
        diff.z = diff.z - copysignf(size_z, diff.z);
    }

    return diff;
}

// Calculate flocking forces for each bird using the spatial grid
__global__ void calculateForces(float separationWeight, float alignmentWeight, float cohesionWeight) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numBirds) {
        Bird& bird = birds[tid];

        float3 separation = make_float3(0.0f, 0.0f, 0.0f);
        float3 alignment = make_float3(0.0f, 0.0f, 0.0f);
        float3 cohesion = make_float3(0.0f, 0.0f, 0.0f);
        int separationCount = 0;
        int alignmentCount = 0;
        int cohesionCount = 0;

        // Get current bird's cell
        int3 cellCoords = getCellCoords(
            bird.position,
            grid.minBounds,
            grid.cellSize,
            grid.gridSizeX,
            grid.gridSizeY,
            grid.gridSizeZ
        );

        // Check birds in current cell and neighboring cells
        for (int offsetZ = -1; offsetZ <= 1; offsetZ++) {
            for (int offsetY = -1; offsetY <= 1; offsetY++) {
                for (int offsetX = -1; offsetX <= 1; offsetX++) {
                    int3 neighborCellCoords = make_int3(
                        cellCoords.x + offsetX,
                        cellCoords.y + offsetY,
                        cellCoords.z + offsetZ
                    );

                    // Skip out-of-bounds cells
                    if (neighborCellCoords.x < 0 || neighborCellCoords.x >= grid.gridSizeX ||
                        neighborCellCoords.y < 0 || neighborCellCoords.y >= grid.gridSizeY ||
                        neighborCellCoords.z < 0 || neighborCellCoords.z >= grid.gridSizeZ) {
                        continue;
                    }

                    int neighborCellIndex = getCellIndex(
                        neighborCellCoords,
                        grid.gridSizeX,
                        grid.gridSizeY
                    );

                    int start = grid.cellStartIndices[neighborCellIndex];
                    int end = grid.cellEndIndices[neighborCellIndex];

                    if (start == -1) continue; // Empty cell

                    // Check all birds in this cell
                    for (int j = start; j <= end; j++) {
                        int otherBirdIndex = grid.particleIndices[j];

                        if (otherBirdIndex == tid) continue; // Skip self

                        Bird& other = birds[otherBirdIndex];
                        float3 diff = calculateWrappedDistance(bird.position, other.position);
                        float dist_sq = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;

                        // Separation
                        if (dist_sq > EPSILON && dist_sq < (PERCEPTION_RADIUS * SEPARATION_RADIUS_FACTOR) * (PERCEPTION_RADIUS * SEPARATION_RADIUS_FACTOR)) {
                            float3 repulse = make_float3(diff.x / dist_sq, diff.y / dist_sq, diff.z / dist_sq);
                            separation.x += repulse.x;
                            separation.y += repulse.y;
                            separation.z += repulse.z;
                            separationCount++;
                        }

                        // For both alignment and cohesion we use the full perception radius
                        if (dist_sq < PERCEPTION_RADIUS * PERCEPTION_RADIUS) {
                            // Alignment
                            alignment.x += other.velocity.x;
                            alignment.y += other.velocity.y;
                            alignment.z += other.velocity.z;
                            alignmentCount++;

                            // Cohesion
                            float3 otherPos = make_float3(
                                bird.position.x - diff.x,
                                bird.position.y - diff.y,
                                bird.position.z - diff.z
                            );
                            cohesion.x += otherPos.x;
                            cohesion.y += otherPos.y;
                            cohesion.z += otherPos.z;
                            cohesionCount++;
                        }
                    }
                }
            }
        }

        // Compute final forces with weights
        float3 separationForce = make_float3(0.0f, 0.0f, 0.0f);
        float3 alignmentForce = make_float3(0.0f, 0.0f, 0.0f);
        float3 cohesionForce = make_float3(0.0f, 0.0f, 0.0f);

        // Finalize separation force
        if (separationCount > 0) {
            separation.x /= separationCount;
            separation.y /= separationCount;
            separation.z /= separationCount;

            float len = sqrtf(separation.x * separation.x + separation.y * separation.y + separation.z * separation.z);
            if (len > EPSILON) {
                separation.x /= len;
                separation.y /= len;
                separation.z /= len;
            }

            separationForce = make_float3(
                separation.x * MAX_FORCE,
                separation.y * MAX_FORCE,
                separation.z * MAX_FORCE
            );
        }

        // Finalize alignment force
        if (alignmentCount > 0) {
            alignment.x /= alignmentCount;
            alignment.y /= alignmentCount;
            alignment.z /= alignmentCount;

            float3 steer = make_float3(
                alignment.x - bird.velocity.x,
                alignment.y - bird.velocity.y,
                alignment.z - bird.velocity.z
            );

            float len = sqrtf(steer.x * steer.x + steer.y * steer.y + steer.z * steer.z);
            if (len > MAX_FORCE && len > EPSILON) {
                steer.x = (steer.x / len) * MAX_FORCE;
                steer.y = (steer.y / len) * MAX_FORCE;
                steer.z = (steer.z / len) * MAX_FORCE;
            }

            alignmentForce = steer;
        }

        // Finalize cohesion force
        if (cohesionCount > 0) {
            cohesion.x /= cohesionCount;
            cohesion.y /= cohesionCount;
            cohesion.z /= cohesionCount;

            float3 desired = make_float3(
                cohesion.x - bird.position.x,
                cohesion.y - bird.position.y,
                cohesion.z - bird.position.z
            );

            float len = sqrtf(desired.x * desired.x + desired.y * desired.y + desired.z * desired.z);
            if (len > EPSILON) {
                desired.x = (desired.x / len) * MAX_SPEED;
                desired.y = (desired.y / len) * MAX_SPEED;
                desired.z = (desired.z / len) * MAX_SPEED;
            }

            float3 steer = make_float3(
                desired.x - bird.velocity.x,
                desired.y - bird.velocity.y,
                desired.z - bird.velocity.z
            );

            len = sqrtf(steer.x * steer.x + steer.y * steer.y + steer.z * steer.z);
            if (len > MAX_FORCE && len > EPSILON) {
                steer.x = (steer.x / len) * MAX_FORCE;
                steer.y = (steer.y / len) * MAX_FORCE;
                steer.z = (steer.z / len) * MAX_FORCE;
            }

            cohesionForce = steer;
        }

        // Apply weights to forces
        separationForce.x *= separationWeight;
        separationForce.y *= separationWeight;
        separationForce.z *= separationWeight;

        alignmentForce.x *= alignmentWeight;
        alignmentForce.y *= alignmentWeight;
        alignmentForce.z *= alignmentWeight;

        cohesionForce.x *= cohesionWeight;
        cohesionForce.y *= cohesionWeight;
        cohesionForce.z *= cohesionWeight;

        // Determine dominant force for coloring
        float sepMag = sqrtf(separationForce.x * separationForce.x +
            separationForce.y * separationForce.y +
            separationForce.z * separationForce.z);
        float aliMag = sqrtf(alignmentForce.x * alignmentForce.x +
            alignmentForce.y * alignmentForce.y +
            alignmentForce.z * alignmentForce.z);
        float cohMag = sqrtf(cohesionForce.x * cohesionForce.x +
            cohesionForce.y * cohesionForce.y +
            cohesionForce.z * cohesionForce.z);

        if (sepMag > aliMag && sepMag > cohMag) {
            bird.dominantForce = 0;  // Separation dominant (red)
        }
        else if (aliMag > sepMag && aliMag > cohMag) {
            bird.dominantForce = 1;  // Alignment dominant (blue)
        }
        else {
            bird.dominantForce = 2;  // Cohesion dominant (green)
        }

        // Sum all forces
        bird.acceleration.x = separationForce.x + alignmentForce.x + cohesionForce.x;
        bird.acceleration.y = separationForce.y + alignmentForce.y + cohesionForce.y;
        bird.acceleration.z = separationForce.z + alignmentForce.z + cohesionForce.z;
    }
}

// Update bird positions based on velocity and acceleration
__global__ void updatePositions(float dt) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numBirds) {
        Bird& bird = birds[tid];

        // Update velocity by acceleration
        bird.velocity.x += bird.acceleration.x * dt;
        bird.velocity.y += bird.acceleration.y * dt;
        bird.velocity.z += bird.acceleration.z * dt;

        // Limit speed to maximum
        float speed = sqrtf(bird.velocity.x * bird.velocity.x +
            bird.velocity.y * bird.velocity.y +
            bird.velocity.z * bird.velocity.z);
        if (speed > MAX_SPEED) {
            bird.velocity.x = (bird.velocity.x / speed) * MAX_SPEED;
            bird.velocity.y = (bird.velocity.y / speed) * MAX_SPEED;
            bird.velocity.z = (bird.velocity.z / speed) * MAX_SPEED;
        }

        // Update position by velocity
        bird.position.x += bird.velocity.x * dt;
        bird.position.y += bird.velocity.y * dt;
        bird.position.z += bird.velocity.z * dt;

        // Reset acceleration
        bird.acceleration = make_float3(0.0f, 0.0f, 0.0f);

        // Handle boundary conditions (wraparound)
        if (bird.position.x < minBounds.x) bird.position.x = maxBounds.x;
        if (bird.position.y < minBounds.y) bird.position.y = maxBounds.y;
        if (bird.position.z < minBounds.z) bird.position.z = maxBounds.z;
        if (bird.position.x > maxBounds.x) bird.position.x = minBounds.x;
        if (bird.position.y > maxBounds.y) bird.position.y = minBounds.y;
        if (bird.position.z > maxBounds.z) bird.position.z = minBounds.z;
    }
}

// Ray structure for rendering
struct Ray {
    float3 origin;
    float3 direction;
};

// Render birds using simple ray casting
__global__ void renderBirdsKernel(uchar4* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    // Calculate normalized device coordinates
    float u = (float)x / width * 2.0f - 1.0f;
    float v = (float)(height - y) / height * 2.0f - 1.0f;

    // Maintain aspect ratio
    u *= (float)width / height;

    // Camera setup
    float3 cameraPos = make_float3(0.0f, 0.0f, 120.0f);
    float3 lookAt = make_float3(0.0f, 0.0f, 0.0f);
    float3 up = make_float3(0.0f, 1.0f, 0.0f);

    // Create normalized camera direction vectors
    float3 forward = make_float3(
        lookAt.x - cameraPos.x,
        lookAt.y - cameraPos.y,
        lookAt.z - cameraPos.z
    );
    float len = sqrtf(forward.x * forward.x + forward.y * forward.y + forward.z * forward.z);
    forward.x /= len;
    forward.y /= len;
    forward.z /= len;

    float3 right = make_float3(
        up.y * forward.z - up.z * forward.y,
        up.z * forward.x - up.x * forward.z,
        up.x * forward.y - up.y * forward.x
    );
    len = sqrtf(right.x * right.x + right.y * right.y + right.z * right.z);
    right.x /= len;
    right.y /= len;
    right.z /= len;

    float3 camUp = make_float3(
        forward.y * right.z - forward.z * right.y,
        forward.z * right.x - forward.x * right.z,
        forward.x * right.y - forward.y * right.x
    );

    // Create ray direction
    float3 rayDirection = make_float3(
        forward.x + u * right.x + v * camUp.x,
        forward.y + u * right.y + v * camUp.y,
        forward.z + u * right.z + v * camUp.z
    );
    len = sqrtf(rayDirection.x * rayDirection.x + rayDirection.y * rayDirection.y + rayDirection.z * rayDirection.z);
    rayDirection.x /= len;
    rayDirection.y /= len;
    rayDirection.z /= len;

    Ray ray;
    ray.origin = cameraPos;
    ray.direction = rayDirection;

    // Background color
    uchar4 bgColor = make_uchar4(25, 25, 40, 255);
    output[y * width + x] = bgColor;

    // Render each bird as a simple colored sphere
    const float birdRadius = 1.0f;
    float closestHit = 1e30f;
    int closestBird = -1;

    for (int i = 0; i < numBirds; i++) {
        float3 oc = make_float3(
            ray.origin.x - birds[i].position.x,
            ray.origin.y - birds[i].position.y,
            ray.origin.z - birds[i].position.z
        );

        float a = ray.direction.x * ray.direction.x +
            ray.direction.y * ray.direction.y +
            ray.direction.z * ray.direction.z;
        float b = 2.0f * (oc.x * ray.direction.x +
            oc.y * ray.direction.y +
            oc.z * ray.direction.z);
        float c = oc.x * oc.x + oc.y * oc.y + oc.z * oc.z - birdRadius * birdRadius;

        float discriminant = b * b - 4 * a * c;

        if (discriminant > 0) {
            float temp = (-b - sqrtf(discriminant)) / (2.0f * a);

            if (temp > 0.001f && temp < closestHit) {
                closestHit = temp;
                closestBird = i;
            }
        }
    }

    // If ray intersects with a bird sphere, color it based on its dominant force
    if (closestBird >= 0) {
        uchar4 color;

        switch (birds[closestBird].dominantForce) {
        case 0: // Separation = Red
            color = make_uchar4(230, 50, 50, 255);
            break;
        case 1: // Alignment = Blue
            color = make_uchar4(50, 50, 230, 255);
            break;
        case 2: // Cohesion = Green
            color = make_uchar4(50, 230, 50, 255);
            break;
        default:
            color = make_uchar4(200, 200, 200, 255);
        }

        // Simple lighting
        float3 hitPoint = make_float3(
            ray.origin.x + closestHit * ray.direction.x,
            ray.origin.y + closestHit * ray.direction.y,
            ray.origin.z + closestHit * ray.direction.z
        );

        float3 normal = make_float3(
            hitPoint.x - birds[closestBird].position.x,
            hitPoint.y - birds[closestBird].position.y,
            hitPoint.z - birds[closestBird].position.z
        );

        len = sqrtf(normal.x * normal.x + normal.y * normal.y + normal.z * normal.z);
        normal.x /= len;
        normal.y /= len;
        normal.z /= len;

        float3 lightDir = make_float3(0.5f, 0.5f, 1.0f);
        len = sqrtf(lightDir.x * lightDir.x + lightDir.y * lightDir.y + lightDir.z * lightDir.z);
        lightDir.x /= len;
        lightDir.y /= len;
        lightDir.z /= len;

        float diffuse = normal.x * lightDir.x + normal.y * lightDir.y + normal.z * lightDir.z;
        diffuse = max(0.2f, diffuse); // Ambient + diffuse

        color.x = min(255, (int)(color.x * diffuse));
        color.y = min(255, (int)(color.y * diffuse));
        color.z = min(255, (int)(color.z * diffuse));

        output[y * width + x] = color;
    }
}

// Initialize performance metrics
__global__ void initMetrics() {
    metrics.gridUpdateTime = 0.0f;
    metrics.forceCalculationTime = 0.0f;
    metrics.positionUpdateTime = 0.0f;
    metrics.stepsCompleted = 0;
    metrics.totalTime = 0.0f;
}

// Reset performance metrics
__global__ void resetMetrics() {
    metrics.gridUpdateTime = 0.0f;
    metrics.forceCalculationTime = 0.0f;
    metrics.positionUpdateTime = 0.0f;
    metrics.stepsCompleted = 0;
    metrics.totalTime = 0.0f;
}

// Free simulation resources - internal implementation
void freeCudaResources() {
    if (d_states) {
        hipFree(d_states);
        d_states = nullptr;
    }

    if (d_cellIndices) {
        hipFree(d_cellIndices);
        d_cellIndices = nullptr;
    }

    if (d_particleIndices) {
        hipFree(d_particleIndices);
        d_particleIndices = nullptr;
    }

    if (d_cellStartIndices) {
        hipFree(d_cellStartIndices);
        d_cellStartIndices = nullptr;
    }

    if (d_cellEndIndices) {
        hipFree(d_cellEndIndices);
        d_cellEndIndices = nullptr;
    }

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipEventDestroy(gridStartEvent);
    hipEventDestroy(gridStopEvent);
    hipEventDestroy(forceStartEvent);
    hipEventDestroy(forceStopEvent);
    hipEventDestroy(posStartEvent);
    hipEventDestroy(posStopEvent);
}

// Initialize the simulation
extern "C" void initSimulation(int birdCount, float* minBoundsArray, float* maxBoundsArray) {
    // Set simulation parameters
    hipMemcpyToSymbol(HIP_SYMBOL(numBirds), &birdCount, sizeof(int));

    float3 mins = make_float3(minBoundsArray[0], minBoundsArray[1], minBoundsArray[2]);
    float3 maxs = make_float3(maxBoundsArray[0], maxBoundsArray[1], maxBoundsArray[2]);

    hipMemcpyToSymbol(HIP_SYMBOL(minBounds), &mins, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(maxBounds), &maxs, sizeof(float3));

    // Calculate grid dimensions
    float sizeX = maxs.x - mins.x;
    float sizeY = maxs.y - mins.y;
    float sizeZ = maxs.z - mins.z;

    int gridSizeX = (int)ceilf(sizeX / CELL_SIZE) + 1;
    int gridSizeY = (int)ceilf(sizeY / CELL_SIZE) + 1;
    int gridSizeZ = (int)ceilf(sizeZ / CELL_SIZE) + 1;
    int totalCells = gridSizeX * gridSizeY * gridSizeZ;

    // Initialize spatial grid
    SpatialGrid h_grid;
    h_grid.gridSizeX = gridSizeX;
    h_grid.gridSizeY = gridSizeY;
    h_grid.gridSizeZ = gridSizeZ;
    h_grid.cellSize = CELL_SIZE;
    h_grid.minBounds = mins;

    hipMalloc(&d_cellIndices, birdCount * sizeof(int));
    hipMalloc(&d_particleIndices, birdCount * sizeof(int));
    hipMalloc(&d_cellStartIndices, totalCells * sizeof(int));
    hipMalloc(&d_cellEndIndices, totalCells * sizeof(int));

    h_grid.cells = d_cellIndices;
    h_grid.particleIndices = d_particleIndices;
    h_grid.cellStartIndices = d_cellStartIndices;
    h_grid.cellEndIndices = d_cellEndIndices;

    hipMemcpyToSymbol(HIP_SYMBOL(grid), &h_grid, sizeof(SpatialGrid));

    // Create CUDA events for timing
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&gridStartEvent);
    hipEventCreate(&gridStopEvent);
    hipEventCreate(&forceStartEvent);
    hipEventCreate(&forceStopEvent);
    hipEventCreate(&posStartEvent);
    hipEventCreate(&posStopEvent);

    // Initialize RNG
    hipMalloc(&d_states, birdCount * sizeof(hiprandState));

    // Calculate kernel launch parameters
    dim3 blockSize(256);
    dim3 gridSize((birdCount + blockSize.x - 1) / blockSize.x);
    dim3 cellGridSize((totalCells + blockSize.x - 1) / blockSize.x);

    // Initialize performance metrics
    initMetrics << <1, 1 >> > ();
    hipDeviceSynchronize();

    // Initialize grid
    resetGrid << <cellGridSize, blockSize >> > (d_cellStartIndices, d_cellEndIndices, totalCells);
    hipDeviceSynchronize();

    // Initialize random states and birds
    setupRNG << <gridSize, blockSize >> > (d_states);
    hipDeviceSynchronize();

    initBirds << <gridSize, blockSize >> > (d_states);
    hipDeviceSynchronize();

    printf("Simulation initialized with %d birds and spatial grid of %d x %d x %d cells\n",
        birdCount, gridSizeX, gridSizeY, gridSizeZ);
}

// Update the simulation for one time step
extern "C" void updateSimulation(float dt, float separationWeight, float alignmentWeight, float cohesionWeight) {
    SpatialGrid h_grid;
    hipMemcpyFromSymbol(&h_grid, HIP_SYMBOL(grid), sizeof(SpatialGrid));

    int totalCells = h_grid.gridSizeX * h_grid.gridSizeY * h_grid.gridSizeZ;

    dim3 blockSize(256);
    dim3 gridSize((numBirds + blockSize.x - 1) / blockSize.x);
    dim3 cellGridSize((totalCells + blockSize.x - 1) / blockSize.x);

    float elapsedTime;

    // Start total timing
    hipEventRecord(startEvent);

    // Update spatial grid
    hipEventRecord(gridStartEvent);
    resetGrid << <cellGridSize, blockSize >> > (d_cellStartIndices, d_cellEndIndices, totalCells);
    calculateCellIndices << <gridSize, blockSize >> > (d_particleIndices, d_cellIndices);
    countCellElements << <gridSize, blockSize >> > (d_cellIndices, d_cellStartIndices, d_cellEndIndices);
    hipEventRecord(gridStopEvent);

    // Calculate forces
    hipEventRecord(forceStartEvent);
    calculateForces << <gridSize, blockSize >> > (separationWeight, alignmentWeight, cohesionWeight);
    hipEventRecord(forceStopEvent);

    // Update positions
    hipEventRecord(posStartEvent);
    updatePositions << <gridSize, blockSize >> > (dt);
    hipEventRecord(posStopEvent);

    // End total timing
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);

    // Calculate elapsed times
    hipEventElapsedTime(&elapsedTime, gridStartEvent, gridStopEvent);
    PerformanceMetrics h_metrics;
    hipMemcpyFromSymbol(&h_metrics, HIP_SYMBOL(metrics), sizeof(PerformanceMetrics));
    h_metrics.gridUpdateTime += elapsedTime / 1000.0f;

    hipEventElapsedTime(&elapsedTime, forceStartEvent, forceStopEvent);
    h_metrics.forceCalculationTime += elapsedTime / 1000.0f;

    hipEventElapsedTime(&elapsedTime, posStartEvent, posStopEvent);
    h_metrics.positionUpdateTime += elapsedTime / 1000.0f;

    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    h_metrics.totalTime += elapsedTime / 1000.0f;

    h_metrics.stepsCompleted++;

    hipMemcpyToSymbol(HIP_SYMBOL(metrics), &h_metrics, sizeof(PerformanceMetrics));
}

// Perform benchmark run without visualization
extern "C" void runBenchmark(int birdCount, int steps, float dt, float separationWeight, float alignmentWeight, float cohesionWeight) {
    // Reset performance metrics
    resetMetrics << <1, 1 >> > ();
    hipDeviceSynchronize();

    printf("Running benchmark for %d birds with %d steps...\n", birdCount, steps);

    for (int i = 1; i <= steps; i++) {
        updateSimulation(dt, separationWeight, alignmentWeight, cohesionWeight);

        if (i % 100 == 0 || i == steps) {
            printf("Completed %d steps (%.1f%%)\n", i, (i * 100.0f) / steps);
        }
    }

    // Get final metrics
    PerformanceMetrics h_metrics;
    hipMemcpyFromSymbol(&h_metrics, HIP_SYMBOL(metrics), sizeof(PerformanceMetrics));

    // Report performance
    printf("\n=== Performance Report ===\n");
    printf("Total steps: %d\n", h_metrics.stepsCompleted);
    printf("Total time: %.3f seconds\n", h_metrics.totalTime);
    printf("Steps per second: %.1f\n", h_metrics.stepsCompleted / h_metrics.totalTime);
    printf("Time breakdown:\n");
    printf("  - Spatial grid updates: %.3fs (%.1f%%)\n",
        h_metrics.gridUpdateTime,
        (h_metrics.gridUpdateTime * 100.0f) / h_metrics.totalTime);
    printf("  - Force calculations: %.3fs (%.1f%%)\n",
        h_metrics.forceCalculationTime,
        (h_metrics.forceCalculationTime * 100.0f) / h_metrics.totalTime);
    printf("  - Position updates: %.3fs (%.1f%%)\n",
        h_metrics.positionUpdateTime,
        (h_metrics.positionUpdateTime * 100.0f) / h_metrics.totalTime);

    float overhead = h_metrics.totalTime -
        (h_metrics.gridUpdateTime +
            h_metrics.forceCalculationTime +
            h_metrics.positionUpdateTime);

    printf("  - Other/overhead: %.3fs (%.1f%%)\n",
        overhead,
        (overhead * 100.0f) / h_metrics.totalTime);
    printf("=========================\n");
}

// Perform scaling test with different flock sizes
extern "C" void runScalingTest(int* flockSizes, int numSizes, int steps, float dt, float separationWeight, float alignmentWeight, float cohesionWeight) {
    printf("Running scaling test with various flock sizes...\n");

    for (int s = 0; s < numSizes; s++) {
        int birdCount = flockSizes[s];
        printf("\n=== Testing with %d birds ===\n", birdCount);

        // Reinitialize simulation with new bird count
        float minBoundsArray[3] = { -50.0f, -50.0f, -50.0f };
        float maxBoundsArray[3] = { 50.0f, 50.0f, 50.0f };

        // Free previous resources
        freeCudaResources();

        // Initialize with new count
        initSimulation(birdCount, minBoundsArray, maxBoundsArray);

        // Run benchmark
        runBenchmark(birdCount, steps, dt, separationWeight, alignmentWeight, cohesionWeight);
    }
}

// Render the birds to the output buffer
extern "C" void renderBirds(int width, int height, uchar4* output) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
        (height + blockSize.y - 1) / blockSize.y);

    renderBirdsKernel << <gridSize, blockSize >> > (output, width, height);
}

// Get performance metrics
extern "C" void getPerformanceMetrics(PerformanceMetrics* out_metrics) {
    hipMemcpyFromSymbol(out_metrics, HIP_SYMBOL(metrics), sizeof(PerformanceMetrics));
}

// Free simulation resources - exported function
extern "C" void freeSimulation() {
    freeCudaResources();
}